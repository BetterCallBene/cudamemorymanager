#include "cuda/common.hpp"
#include "HostMemory.hpp"

void TraPla::future::HostMemory::allocate(uint64 size)
{
    CUDA_SAFE_CALL(hipHostMalloc((void**)&_host_ptr, size));  
}

void TraPla::future::HostMemory::deallocate()
{
    CUDA_SAFE_CALL(hipHostFree(_host_ptr));
}